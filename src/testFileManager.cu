#include "hip/hip_runtime.h"
#include "FileManager.hpp"
#include <iostream>
#include <fstream>


int main(int argc, char ** argv)
{
	if (argc < 3)
	{
		std::cerr<<"Error, testFileManger hdf5_file dataset_name"<<std::endl;
		return 0;
	}

	FileManager *  fileManager = new hdf5File(argv[1], argv[2], 4, 9, make_int3(32,32,32), make_int3(2,2,2));

	float * data = new float[36*36*36];

	fileManager->readCube(5500, data);

	for(int i=0;i<(36*36*36); i++)
		std::cout<<data[i]<<std::endl;

	delete fileManager;
	delete[] data;
}
