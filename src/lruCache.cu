#include "hip/hip_runtime.h"
#include "lruCache.hpp"
#include <exception>
#include <iostream>
#include <fstream>
#include <strings.h>

LinkedList::LinkedList(int size)
{
	freePositions 	= size;
	memoryList 	= new NodeLinkedList[size];
	list 		= memoryList;
	last 		= &memoryList[size-1];

	for(int i=0; i<size; i++)
	{
		if (i==0)
		{
			memoryList[i].after 		= &memoryList[i+1];
			memoryList[i].before 		= 0;
			memoryList[i].element 		= i;
			memoryList[i].cubeID 		= 0;
			memoryList[i].references 	= 0;
		}
		else if (i==size-1)
		{
			memoryList[i].after 		= 0;
			memoryList[i].before 		= &memoryList[i-1];
			memoryList[i].element 		= i;
			memoryList[i].cubeID 		= 0;
			memoryList[i].references 	= 0;
		}
		else
		{
			memoryList[i].after 		= &memoryList[i+1];
			memoryList[i].before 		= &memoryList[i-1];
			memoryList[i].element 		= i;
			memoryList[i].cubeID 		= 0;
			memoryList[i].references 	= 0;
		}
	}
}

LinkedList::~LinkedList()
{
	delete[] memoryList;
}


NodeLinkedList * LinkedList::getFirstFreePosition(index_node_t newIDcube, index_node_t * removedIDcube)
{
	if (freePositions > 0)
	{
		NodeLinkedList * first = list;

		// Search first free position
		while(first->references != 0)
		{
			moveToLastPosition(first);
			first = list;
		}

		list = first->after;
		list->before = 0;
		
		first->after  = 0;
		first->before = last;
		
		last->after = first;
		
		last = first;
		*removedIDcube = last->cubeID;
		last->cubeID = newIDcube;

		return first;
	}

	return NULL;
}

NodeLinkedList * LinkedList::moveToLastPosition(NodeLinkedList * node)
{
	if (node->before == 0)
	{
		NodeLinkedList * first = list;

		list = first->after;
		list->before = 0;
		
		first->after  = 0;
		first->before = last;
		
		last->after = first;
		
		last = first;

		return first;
	}
	else if (node->after == 0)
	{
		return node;
	}
	else
	{
		node->before->after = node->after;
		node->after->before = node->before;
		
		last->after = node;
		
		node->before = last;
		node->after  = 0;
		last = node;
		
		return node;
	}
}

void	LinkedList::removeReference(NodeLinkedList * node, int ref)
{
	node->references &= ~(ref);

	if (node->references == 0)
		freePositions++;
}

void 	LinkedList::addReference(NodeLinkedList * node, int ref)
{
	if (node->references == 0)
		freePositions--;

	node->references |= ref;
}


lruCache::lruCache(int p_maxElements, int3 p_cubeDim, int p_cubeInc, int p_levelCube, int p_nLevels)
{
	// cube size
	cubeDim 	= p_cubeDim;
	cubeInc		= make_int3(p_cubeInc,p_cubeInc,p_cubeInc);
	realcubeDim	= p_cubeDim + 2 * p_cubeInc;
	levelCube	= p_levelCube;
	nLevels		= p_nLevels;
	offsetCube	= (cubeDim.x+2*cubeInc.x)*(cubeDim.y+2*cubeInc.y)*(cubeDim.z+2*cubeInc.z);

	// Creating caches
	maxElements	= p_maxElements;
	queuePositions	= new LinkedList(maxElements);

	// Creating mutex needed to synchronization
	lock = new lunchbox::Lock();
}

Cache::Cache(char ** argv, int p_maxElements, int3 p_cubeDim, int p_cubeInc, int p_levelCube, int p_nLevels)
{
	if (strcmp(argv[0], "GPU_FILE") == 0)
	{
		cache = new cache_GPU_File(&argv[1], p_maxElements, p_cubeDim, p_cubeInc, p_levelCube, p_nLevels);
	}
	else
	{
		std::cerr<<"Error: cache options error"<<std::endl;
		throw;
	}
}

Cache::~Cache()
{
	delete cache;
}

int Cache::getCacheLevel()
{
	return cache->getCacheLevel();
}

void Cache::push(visibleCube_t * visibleCubes, int num, int octreeLevel, threadID_t * thread)
{
	// For each visible cube push into the cache
	for(int i=0; i<num; i++)
	{
		cache->push_cube(&visibleCubes[i], octreeLevel, thread);
	}
}

void Cache::pop(visibleCube_t * visibleCubes, int num, int octreeLevel, threadID_t * thread)
{
	// For each visible cube pop out the cache
	for(int i=0; i<num; i++)
	{
		cache->pop_cube(&visibleCubes[i], octreeLevel, thread);
	}
}
