#include "hip/hip_runtime.h"
#include "lruCache.hpp"
#include <Exceptions.hpp>
#include <iostream>
#include <fstream>

LinkedList::LinkedList(int size)
{
	memoryList = new NodeLinkedList[size];
	list = memoryList;
	last = &memoryList[size-1];
	for(int i=0; i<size; i++)
	{
		if (i==0)
		{
			memoryList[i].after = &memoryList[i+1];
			memoryList[i].before = 0;
			memoryList[i].element = i;
			memoryList[i].cubeID = 0;
		}
		else if (i==size-1)
		{
			memoryList[i].after = 0;
			memoryList[i].before = &memoryList[i-1];
			memoryList[i].element = i;
			memoryList[i].cubeID = 0;
		}
		else
		{
			memoryList[i].after = &memoryList[i+1];
			memoryList[i].before = &memoryList[i-1];
			memoryList[i].element = i;
			memoryList[i].cubeID = 0;
		}
	}
}

LinkedList::~LinkedList()
{
	delete[] memoryList;
}


NodeLinkedList * LinkedList::getFromFirstPosition(index_node_t newIDcube, index_node_t * removedIDcube)
{
	NodeLinkedList * first = list;

	list = first->after;
	list->before = 0;
	
	first->after  = 0;
	first->before = last;
	
	last->after = first;
	
	last = first;
	*removedIDcube = last->cubeID;
	last->cubeID = newIDcube;

	return first;

}

NodeLinkedList * LinkedList::moveToLastPosition(NodeLinkedList * node)
{
	if (node->before == 0)
	{
		NodeLinkedList * first = list;

		list = first->after;
		list->before = 0;
		
		first->after  = 0;
		first->before = last;
		
		last->after = first;
		
		last = first;

		return first;
	}
	else if (node->after == 0)
	{
		return node;
	}
	else
	{
		node->before->after = node->after;
		node->after->before = node->before;
		
		last->after = node;
		
		node->before = last;
		node->after  = 0;
		last = node;
		
		return node;
	}
}


lruCache::lruCache(char ** argv, int p_maxElementsGPU, int3 p_cubeDim, int p_cubeInc, int p_levelCube, int p_levelsOctree, int p_nLevels, int p_maxElementsCPU)
{
	maxElementsGPU 	= p_maxElementsGPU;
	maxElementsCPU 	= p_maxElementsCPU;
	cubeDim 	= p_cubeDim;
	cubeInc		= make_int3(p_cubeInc,p_cubeInc,p_cubeInc);
	realcubeDim	= p_cubeDim + 2 * p_cubeInc;
	levelCube	= p_levelCube;
	levelOctree	= p_levelsOctree;
	nLevels		= p_nLevels;
	offsetCube	= (cubeDim.x+2*cubeInc.x)*(cubeDim.y+2*cubeInc.y)*(cubeDim.z+2*cubeInc.z);
	queuePositionsGPU	= new LinkedList(maxElementsGPU);
	queuePositionsCPU	= new LinkedList(maxElementsCPU);

	std::cerr<<"Creating cache in GPU: "<< maxElementsGPU*offsetCube*sizeof(float)/1024/1024<<" MB: "<< hipGetErrorString(hipMalloc((void**)&cacheDataGPU, maxElementsGPU*offsetCube*sizeof(float)))<<std::endl;
	std::cerr<<"Creating cache in CPU: "<< maxElementsCPU*offsetCube*sizeof(float)/1024/1024<<" MB: "<< hipGetErrorString(hipHostAlloc((void**)&cacheDataCPU, maxElementsCPU*offsetCube*sizeof(float),hipHostMallocDefault))<<std::endl;

	// Open File
	fileManager = OpenFile(argv, levelCube, nLevels, cubeDim, cubeInc);

	if(pthread_mutex_init(&mutex, NULL))
    	{
        	std::cerr<<"Unable to initialize a mutex"<<std::endl;
        	throw excepGen;
    	}	
}

lruCache::~lruCache()
{
	pthread_mutex_destroy(&mutex);

	delete queuePositionsGPU;
	delete queuePositionsCPU;
	delete fileManager;
	hipFree(cacheDataGPU);
	hipHostFree(cacheDataCPU);
}

void lruCache::push(visibleCube_t * visibleCubes, int num, threadID_t * thread)
{
	for(int i=0; i<num; i++)
	{
		pthread_mutex_lock(&mutex);

		push_cube(&visibleCubes[i], thread);

		pthread_mutex_unlock(&mutex);
	}
}

void lruCache::pop(visibleCube_t * visibleCubes, int num, threadID_t * thread)
{
	for(int i=0; i<num; i++)
	{
		pthread_mutex_lock(&mutex);

		pop_cube(&visibleCubes[i], thread);

		pthread_mutex_unlock(&mutex);
	}
}

void lruCache::push_cube(visibleCube_t * cube, threadID_t * thread)
{
	return;
}

void lruCache::pop_cube(visibleCube_t * cube, threadID_t * thread)
{
	return;
}
