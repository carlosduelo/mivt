#include "hip/hip_runtime.h"
#include "threadMaster.hpp"
#include "FreeImage.h"
#include <exception>
#include <iostream>
#include <fstream>

int main(int argc, char ** argv)
{
	if (argc < 3)
	{
		std::cerr<<"Error, testFileManger file_type file [dataset_name]"<<std::endl;
		return 0;
	}

	initParams_masterWorker_t params;

	// Workers
	params.numDevices	= 1;
	params.numWorkers[0]	= 1;
	params.numWorkers[1]	= 2;
	params.numWorkers[2]	= 2;
	params.deviceID[0]	= 0;
	params.deviceID[1]	= 1;
	params.deviceID[2]	= 2;

	// Cache
	params.maxElementsCache[0]	= 100;
	params.maxElementsCache[1]	= 100;
	params.maxElementsCache[2]	= 100;
	params.cubeInc			= 2;
	params.cubeDim			= make_int3(32,32,32);
	params.levelCube		= 4;

	// Octree
	params.maxLevelOctree	= 9;

	// ray caster
	params.rayCasterOptions.ligth_position = make_float3(512.0f, 512.0f, 512.0f);

	// Camera
	params.displayOptions.height		= 512;
	params.displayOptions.width		= 512;
	params.displayOptions.distance		= 50.0f;
	params.displayOptions.fov_H		= 30.0f;
	params.displayOptions.fov_W		= 30.0f;
	params.displayOptions.numRayPixel	= 1;
	params.displayOptions.tileDim		= make_int2(32,32);
	params.displayOptions.position		= make_float3(128.0f, 128.0f, 512.0f);

	threadMaster * mivt = new threadMaster(&argv[1], &params);

	mivt->increaseLevelOctree();
	mivt->decreaseLevelOctree();

	float * buffer = 0;
	std::cerr<<"Allocating pixel buffer: ";
	if (hipSuccess != hipHostMalloc((void**)&buffer, 3*params.displayOptions.height*params.displayOptions.width*sizeof(float)))
	{
		std::cerr<<"Fail"<<std::endl;
		throw;
	}
	else
		std::cerr<<"Ok"<<std::endl;

	FreeImage_Initialise();
	FIBITMAP * bitmap = FreeImage_Allocate(params.displayOptions.width, params.displayOptions.height, 24);
	RGBQUAD color;


	mivt->createFrame(buffer);

#if 1
	for(int i=0; i<params.displayOptions.height; i++)
		for(int j=0; j<params.displayOptions.width; j++)
                {
			int id = i*params.displayOptions.width + j;
			color.rgbRed 	= buffer[id*3]*255;
			color.rgbGreen 	= buffer[id*3+1]*255;
			color.rgbBlue 	= buffer[id*3+2]*255;
			FreeImage_SetPixelColor(bitmap, j, i, &color);
		}

	std::stringstream name;
        name<<"prueba"<<0<<".png";
        FreeImage_Save(FIF_PNG, bitmap, name.str().c_str(), 0);
#endif

	FreeImage_DeInitialise();
	delete mivt;
	hipHostFree(buffer);
}
